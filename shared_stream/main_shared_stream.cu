#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <iostream>
#include <string>

#define taille_stream 2
std::size_t taille_rgb = 0;
std::size_t one_line_rgb = 0;


__global__ void blur2D(const unsigned char* rgb_in, unsigned char* rgb_out_blur, int rows, int cols) {
    auto col = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_blur2D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        for (int rgb = 0; rgb < 3; ++rgb) {
            unsigned char hg = rgb_in[3 * ((row - 1) * cols + col - 1) + rgb];
            unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
            unsigned char hd = rgb_in[3 * ((row - 1) * cols + col + 1) + rgb];
            unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
            unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
            unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
            unsigned char bg = rgb_in[3 * ((row + 1) * cols + col - 1) + rgb];
            unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
            unsigned char bd = rgb_in[3 * ((row + 1) * cols + col + 1) + rgb];

            sh_blur2D[3 * (lrow * blockDim.x + lcol) + rgb] = (hg + h + hd + g + c + d + bg + b + bd) / 9;
            rgb_out_blur[3 * (row * cols + col) + rgb] = sh_blur2D[3 * (lrow * blockDim.x + lcol) + rgb];
        }
    }
}

__global__ void sharpen2D(const unsigned char* rgb_in, unsigned char* rgb_out_sharpen, int rows, int cols) {
    auto col = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_sharpen_2D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        for (int rgb = 0; rgb < 3; ++rgb) {
            unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
            unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
            unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
            unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
            unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
            int somme = (-3 * (h + g + d + b) + 21 * c) / 9;

            if (somme > 255) somme = 255;
            if (somme < 0) somme = 0;

            sh_sharpen_2D[3 * (lrow * blockDim.x + lcol) + rgb] = somme;
            rgb_out_sharpen[3 * (row * cols + col) + rgb] = sh_sharpen_2D[3 * (lrow * blockDim.x + lcol) + rgb];
        }
    }
}

__global__ void edge_detect2D(const unsigned char* rgb_in, unsigned char* rgb_out_edge_detect, int rows, int cols) {
    auto col = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_edge_detect_2D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        for (int rgb = 0; rgb < 3; ++rgb) {
            unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
            unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
            unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
            unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
            unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
            int somme = (9 * (h + g + d + b) - 36 * c) / 9;

            if (somme > 255) somme = 255;
            if (somme < 0) somme = 0;

            sh_edge_detect_2D[3 * (lrow * blockDim.x + lcol) + rgb] = somme;
            rgb_out_edge_detect[3 * (row * cols + col) + rgb] = sh_edge_detect_2D[3 * (lrow * blockDim.x + lcol) + rgb];
        }
    }
}


__global__ void blur3D(const unsigned char* rgb_in, unsigned char* rgb_out_blur, int rows, int cols) {
    auto col = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y
    auto rgb = threadIdx.z;

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_blur3D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        unsigned char hg = rgb_in[3 * ((row - 1) * cols + col - 1) + rgb];
        unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
        unsigned char hd = rgb_in[3 * ((row - 1) * cols + col + 1) + rgb];
        unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
        unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
        unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
        unsigned char bg = rgb_in[3 * ((row + 1) * cols + col - 1) + rgb];
        unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
        unsigned char bd = rgb_in[3 * ((row + 1) * cols + col + 1) + rgb];

        sh_blur3D[3*(lrow * blockDim.x + lcol) + rgb] = (hg + h + hd + g + c + d + bg + b + bd) / 9;
        rgb_out_blur[3 * (row * cols + col) + rgb] = sh_blur3D[3 * (lrow * blockDim.x + lcol) + rgb];
    }
}

__global__ void sharpen3D(const unsigned char* rgb_in, unsigned char* rgb_out_sharpen, int rows, int cols) {
    auto col = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y
    auto rgb = threadIdx.z;

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_sharpen3D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
        unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
        unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
        unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
        unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
        int somme = (-3 * (h + g + d + b) + 21 * c) / 9;

        if (somme > 255) somme = 255;
        if (somme < 0) somme = 0;

        sh_sharpen3D[3 * (lrow * blockDim.x + lcol) + rgb] = somme;
        rgb_out_sharpen[3 * (row * cols + col) + rgb] = sh_sharpen3D[3 * (lrow * blockDim.x + lcol) + rgb];
    }
}

__global__ void edge_detect3D(const unsigned char* rgb_in, unsigned char* rgb_out_edge_detect, int rows, int cols) {
    auto col = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y
    auto rgb = threadIdx.z;

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_edge_detect3D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
        unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
        unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
        unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
        unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
        int somme = (9 * (h + g + d + b) - 36 * c) / 9;

        if (somme > 255) somme = 255;
        if (somme < 0) somme = 0;

        sh_edge_detect3D[3*(lrow * blockDim.x + lcol) + rgb] = somme;
        rgb_out_edge_detect[3 * (row * cols + col) + rgb] = sh_edge_detect3D[3*(lrow * blockDim.x + lcol) + rgb];
    }
}


__global__ void blur_edge_detect2D(const unsigned char * rgb_in, unsigned char * rgb_out_edge_detect, std::size_t rows, std::size_t cols) {
    auto col = blockIdx.x * (blockDim.x - 2) + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * (blockDim.y - 2) + threadIdx.y; //pos de la couleur sur y

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_blur_edge_detect2D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        for (int rgb = 0; rgb < 3; ++rgb) {
            unsigned char hg = rgb_in[3 * ((row - 1) * cols + col - 1) + rgb];
            unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
            unsigned char hd = rgb_in[3 * ((row - 1) * cols + col + 1) + rgb];
            unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
            unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
            unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
            unsigned char bg = rgb_in[3 * ((row + 1) * cols + col - 1) + rgb];
            unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
            unsigned char bd = rgb_in[3 * ((row + 1) * cols + col + 1) + rgb];

            sh_blur_edge_detect2D[3 * (lrow * blockDim.x + lcol) + rgb] = (hg + h + hd + g + c + d + bg + b + bd) / 9;
        }
    } else {
        for (int rgb = 0; rgb < 3; ++rgb) {
            sh_blur_edge_detect2D[3 * (lrow * blockDim.x + lcol) + rgb] = 0;
        }
    }

    __syncthreads();

    auto ww = blockDim.x;

    if (lcol > 0 && lcol < (blockDim.x - 1) && lrow > 0 && lrow < (blockDim.y - 1) &&
        row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        for (int rgb = 0; rgb < 3; ++rgb) {
            unsigned char h = sh_blur_edge_detect2D[3 * ((lrow - 1) * ww + lcol) + rgb];
            unsigned char g = sh_blur_edge_detect2D[3 * (lrow * ww + lcol - 1) + rgb];
            unsigned char c = sh_blur_edge_detect2D[3 * (lrow * ww + lcol) + rgb];
            unsigned char d = sh_blur_edge_detect2D[3 * (lrow * ww + lcol + 1) + rgb];
            unsigned char b = sh_blur_edge_detect2D[3 * ((lrow + 1) * ww + lcol) + rgb];
            int somme = (9 * (h + g + d + b) - 36 * c) / 9;

            if (somme > 255) somme = 255;
            if (somme < 0) somme = 0;

            rgb_out_edge_detect[3 * (row * cols + col) + rgb] = somme;
        }
    }
}

__global__ void edge_detect_blur2D(const unsigned char * rgb_in, unsigned char * rgb_out_edge_detect, std::size_t rows, std::size_t cols) {
    auto col = blockIdx.x * (blockDim.x - 2) + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * (blockDim.y - 2) + threadIdx.y; //pos de la couleur sur y

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_edge_detect_blur2D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        for (int rgb = 0; rgb < 3; ++rgb) {
            unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
            unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
            unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
            unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
            unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
            int somme = (9 * (h + g + d + b) - 36 * c) / 9;

            if (somme > 255) somme = 255;
            if (somme < 0) somme = 0;

            sh_edge_detect_blur2D[3 * (lrow * blockDim.x + lcol) + rgb] = somme;
        }
    } else {
        for (int rgb = 0; rgb < 3; ++rgb) {
            sh_edge_detect_blur2D[3 * (lrow * blockDim.x + lcol) + rgb] = 0;
        }
    }

    __syncthreads();

    auto ww = blockDim.x;

    if (lcol > 0 && lcol < (blockDim.x - 1) && lrow > 0 && lrow < (blockDim.y - 1) &&
        row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        for (int rgb = 0; rgb < 3; ++rgb) {
            unsigned char hg = sh_edge_detect_blur2D[3 * ((lrow - 1) * ww + lcol - 1) + rgb];
            unsigned char h = sh_edge_detect_blur2D[3 * ((lrow - 1) * ww + lcol) + rgb];
            unsigned char hd = sh_edge_detect_blur2D[3 * ((lrow - 1) * ww + lcol + 1) + rgb];
            unsigned char g = sh_edge_detect_blur2D[3 * (lrow * ww + lcol - 1) + rgb];
            unsigned char c = sh_edge_detect_blur2D[3 * (lrow * ww + lcol) + rgb];
            unsigned char d = sh_edge_detect_blur2D[3 * (lrow * ww + lcol + 1) + rgb];
            unsigned char bg = sh_edge_detect_blur2D[3 * ((lrow + 1) * ww + lcol - 1) + rgb];
            unsigned char b = sh_edge_detect_blur2D[3 * ((lrow + 1) * ww + lcol) + rgb];
            unsigned char bd = sh_edge_detect_blur2D[3 * ((lrow + 1) * ww + lcol + 1) + rgb];

            rgb_out_edge_detect[3 * (row * cols + col) + rgb] = (hg + h + hd + g + c + d + bg + b + bd) / 9;
        }
    }
}


__global__ void blur_edge_detect3D(const unsigned char * rgb_in, unsigned char * rgb_out_blur_edge_detect, std::size_t rows, std::size_t cols) {
    auto col = blockIdx.x * (blockDim.x - 2) + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * (blockDim.y - 2) + threadIdx.y; //pos de la couleur sur y
    auto rgb = threadIdx.z;

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_blur_edge_detect3D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        unsigned char hg = rgb_in[3 * ((row - 1) * cols + col - 1) + rgb];
        unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
        unsigned char hd = rgb_in[3 * ((row - 1) * cols + col + 1) + rgb];
        unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
        unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
        unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
        unsigned char bg = rgb_in[3 * ((row + 1) * cols + col - 1) + rgb];
        unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
        unsigned char bd = rgb_in[3 * ((row + 1) * cols + col + 1) + rgb];

        sh_blur_edge_detect3D[3 * (lrow * blockDim.x + lcol) + rgb] = (hg + h + hd + g + c + d + bg + b + bd) / 9;
    } else {
        sh_blur_edge_detect3D[3 * (lrow * blockDim.x + lcol) + rgb] = 0;
    }

    __syncthreads();

    auto ww = blockDim.x;

    if (lcol > 0 && lcol < (blockDim.x - 1) && lrow > 0 && lrow < (blockDim.y - 1) &&
        row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        unsigned char h = sh_blur_edge_detect3D[3 * ((lrow - 1) * ww + lcol) + rgb];
        unsigned char g = sh_blur_edge_detect3D[3 * (lrow * ww + lcol - 1) + rgb];
        unsigned char c = sh_blur_edge_detect3D[3 * (lrow * ww + lcol) + rgb];
        unsigned char d = sh_blur_edge_detect3D[3 * (lrow * ww + lcol + 1) + rgb];
        unsigned char b = sh_blur_edge_detect3D[3 * ((lrow + 1) * ww + lcol) + rgb];
        int somme = (9 * (h + g + d + b) - 36 * c) / 9;

        if (somme > 255) somme = 255;
        if (somme < 0) somme = 0;

        rgb_out_blur_edge_detect[3 * (row * cols + col) + rgb] = somme;
    }
}

__global__ void edge_detect_blur3D(const unsigned char * rgb_in, unsigned char * rgb_out_edge_detect_blur, std::size_t rows, std::size_t cols) {
    auto col = blockIdx.x * (blockDim.x - 2) + threadIdx.x; //pos de la couleur sur x
    auto row = blockIdx.y * (blockDim.y - 2) + threadIdx.y; //pos de la couleur sur y
    auto rgb = threadIdx.z;

    auto lcol = threadIdx.x;
    auto lrow = threadIdx.y;

    extern __shared__ unsigned char sh_edge_detect_blur3D[];

    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
        unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
        unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
        unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
        unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
        int somme = (9 * (h + g + d + b) - 36 * c) / 9;

        if (somme > 255) somme = 255;
        if (somme < 0) somme = 0;

        sh_edge_detect_blur3D[3 * (lrow * blockDim.x + lcol) + rgb] = somme;
    } else {
        sh_edge_detect_blur3D[3 * (lrow * blockDim.x + lcol) + rgb] = 0;
    }

    __syncthreads();

    auto ww = blockDim.x;

    if (lcol > 0 && lcol < (blockDim.x - 1) && lrow > 0 && lrow < (blockDim.y - 1) &&
        row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        unsigned char hg = sh_edge_detect_blur3D[3 * ((lrow - 1) * ww + lcol - 1) + rgb];
        unsigned char h = sh_edge_detect_blur3D[3 * ((lrow - 1) * ww + lcol) + rgb];
        unsigned char hd = sh_edge_detect_blur3D[3 * ((lrow - 1) * ww + lcol + 1) + rgb];
        unsigned char g = sh_edge_detect_blur3D[3 * (lrow * ww + lcol - 1) + rgb];
        unsigned char c = sh_edge_detect_blur3D[3 * (lrow * ww + lcol) + rgb];
        unsigned char d = sh_edge_detect_blur3D[3 * (lrow * ww + lcol + 1) + rgb];
        unsigned char bg = sh_edge_detect_blur3D[3 * ((lrow + 1) * ww + lcol - 1) + rgb];
        unsigned char b = sh_edge_detect_blur3D[3 * ((lrow + 1) * ww + lcol) + rgb];
        unsigned char bd = sh_edge_detect_blur3D[3 * ((lrow + 1) * ww + lcol + 1) + rgb];

        rgb_out_edge_detect_blur[3 * (row * cols + col) + rgb] = (hg + h + hd + g + c + d + bg + b + bd) / 9;
    }
}


void main_blur(const dim3 grid, const dim3 block, const unsigned int shared, const hipStream_t* streams, const unsigned char* rgb_in,
               unsigned char* rgb_out_blur, int rows, int cols) {
    // Debut de chrono
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Appel kernel
    if (block.z == 1) {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            blur2D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                    rgb_out_blur + decalage, row_stream, cols);
        }
    } else {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            blur3D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage, rgb_out_blur + decalage,
                    row_stream, cols);
        }
    }

    // Fin de chrono
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "blur_stream_" << block.x << "-" << block.y << "-" << block.z << ": " << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void main_sharpen(const dim3 grid, const dim3 block, const unsigned int shared, const hipStream_t* streams, const unsigned char* rgb_in,
                  unsigned char* rgb_out_sharpen, int rows, int cols) {
    // Debut de chrono
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Appel kernel
    if (block.z == 1) {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            sharpen2D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                    rgb_out_sharpen + decalage, row_stream, cols);
        }
    } else {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            sharpen3D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage, rgb_out_sharpen + decalage,
                    row_stream, cols);
        }
    }

    // Fin de chrono
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "sharpen_stream_" << block.x << "-" << block.y << "-" << block.z << ": " << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void main_edge_detect(const dim3 grid, const dim3 block, const unsigned int shared, const hipStream_t* streams, const unsigned char* rgb_in,
                      unsigned char* rgb_out_edge_detect, int rows, int cols) {
    // Debut de chrono
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Appel kernel
    if (block.z == 1) {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            edge_detect2D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                    rgb_out_edge_detect + decalage, row_stream, cols);
        }
    } else {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            edge_detect3D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                    rgb_out_edge_detect + decalage, row_stream, cols);
        }
    }

    // Fin de chrono
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "edge_detect_stream_" << block.x << "-" << block.y << "-" << block.z << ": " << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void main_blur_edge_detect(const dim3 grid, const dim3 block, const unsigned int shared, const hipStream_t* streams, const unsigned char* rgb_in,
        unsigned char* rgb_tmp_blur_edge_detect, unsigned char* rgb_out_blur_edge_detect, int rows, int cols) {
    // Debut de chrono
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Appel kernel
    if (block.z == 1) {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            blur2D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                    rgb_tmp_blur_edge_detect + decalage, row_stream, cols);
            edge_detect2D<<< grid, block, shared, streams[i] >>>(rgb_tmp_blur_edge_detect + decalage,
                    rgb_out_blur_edge_detect + decalage, row_stream, cols);
        }
    } else {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            blur3D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                    rgb_tmp_blur_edge_detect + decalage, row_stream, cols);
            edge_detect3D<<< grid, block, shared, streams[i] >>>(rgb_tmp_blur_edge_detect + decalage,
                    rgb_out_blur_edge_detect + decalage, row_stream, cols);
        }
    }

    // Fin de chrono
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "blur_edge_detect_stream_" << block.x << "-" << block.y << "-" << block.z << ": " << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void main_edge_detect_blur(const dim3 grid, const dim3 block, const unsigned int shared, const hipStream_t* streams, const unsigned char* rgb_in,
                           unsigned char* rgb_tmp_blur_edge_detect, unsigned char* rgb_out_blur_edge_detect, int rows, int cols) {
    // Debut de chrono
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Appel kernel
    if (block.z == 1) {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            edge_detect2D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                                                     rgb_tmp_blur_edge_detect + decalage, row_stream, cols);
            blur2D<<< grid, block, shared, streams[i] >>>(rgb_tmp_blur_edge_detect + decalage,
                                                            rgb_out_blur_edge_detect + decalage, row_stream, cols);
        }
    } else {
        for (std::size_t i = 0; i < taille_stream; ++i) {
            int row_stream = (int) (rows / taille_stream) + ((i == 0 || i == taille_stream - 1) ? 1 : 2);
            std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
            edge_detect3D<<< grid, block, shared, streams[i] >>>(rgb_in + decalage,
                                                     rgb_tmp_blur_edge_detect + decalage, row_stream, cols);
            blur3D<<< grid, block, shared, streams[i] >>>(rgb_tmp_blur_edge_detect + decalage,
                                                            rgb_out_blur_edge_detect + decalage, row_stream, cols);
        }
    }

    // Fin de chrono
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "edge_detect_blur_stream_" << block.x << "-" << block.y << "-" << block.z << ": " << elapsedTime << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


int main(int argc, char *argv[])
{
    // Declarations
    hipError_t err;

    std::string filename = std::string(argv[1]) + std::string(".") + std::string(argv[2]);
    std::string out(argv[1]);
    if (out == "in") {
        out = std::string("out");
    }

    cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED);
    unsigned char* rgb = m_in.data;
    int rows = m_in.rows;
    int cols = m_in.cols;

    taille_rgb = 3 * rows * cols;

    std::vector<unsigned char> g_blur(taille_rgb);

    std::vector<unsigned char> g_sharpen(taille_rgb);
    std::vector<unsigned char> g_edge_detect(taille_rgb);

    std::vector<unsigned char> g_blur_edge_detect(taille_rgb);
    std::vector<unsigned char> g_edge_detect_blur(taille_rgb);

    cv::Mat m_out_blur(rows, cols, CV_8UC3, g_blur.data());

    cv::Mat m_out_sharpen(rows, cols, CV_8UC3, g_sharpen.data());
    cv::Mat m_out_edge_detect(rows, cols, CV_8UC3, g_edge_detect.data());

    cv::Mat m_out_blur_edge_detect(rows, cols, CV_8UC3, g_blur_edge_detect.data());
    cv::Mat m_out_edge_detect_blur(rows, cols, CV_8UC3, g_edge_detect_blur.data());

    unsigned char* rgb_in = nullptr;

    unsigned char* rgb_out_blur = nullptr;
    unsigned char* rgb_out_sharpen = nullptr;
    unsigned char* rgb_out_edge_detect = nullptr;

    unsigned char* rgb_tmp_blur_edge_detect = nullptr;
    unsigned char* rgb_tmp_edge_detect_blur = nullptr;
    unsigned char* rgb_out_blur_edge_detect = nullptr;
    unsigned char* rgb_out_edge_detect_blur = nullptr;

    // Init donnes kernel
    err = hipMalloc(&rgb_in, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    err = hipMalloc(&rgb_out_blur, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMalloc(&rgb_out_sharpen, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMalloc(&rgb_out_edge_detect, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    err = hipMalloc(&rgb_tmp_blur_edge_detect, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMalloc(&rgb_tmp_edge_detect_blur, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMalloc(&rgb_out_blur_edge_detect, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMalloc(&rgb_out_edge_detect_blur, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    hipStream_t streams[taille_stream];
    for (std::size_t i = 0; i < taille_stream; ++i) {
        hipStreamCreate(&streams[i]);
    }

    one_line_rgb = 3 * cols;

    for (std::size_t i = 0; i < taille_stream; ++i) {
        std::size_t decalage = i * taille_rgb / taille_stream - (i == 0 ? 0 : one_line_rgb);
        std::size_t count = taille_rgb / taille_stream + ((i == 0 || i == taille_stream - 1) ? one_line_rgb : 2 * one_line_rgb);
        err = hipMemcpyAsync(rgb_in + decalage,rgb + decalage, count, hipMemcpyHostToDevice, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    }

    /////////////////////////////////////////////////////////////////
    ///////////////////// block 32 32 ///////////////////////////////
    /////////////////////////////////////////////////////////////////

    dim3 block_32_32(32, 32 / taille_stream); //nb de thread par bloc, max 1024
    dim3 grid_32_32(((cols - 1) / block_32_32.x + 1), (((rows + (taille_stream - 1) * 2) / taille_stream - 1) / block_32_32.y + 1)); // nb de block
    unsigned int shared = 3 * block_32_32.x * block_32_32.y;

    // Execution
    main_blur(grid_32_32, block_32_32, shared, streams, rgb_in, rgb_out_blur, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    main_sharpen(grid_32_32, block_32_32, shared, streams, rgb_in, rgb_out_sharpen, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    main_edge_detect(grid_32_32, block_32_32, shared, streams, rgb_in, rgb_out_edge_detect, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    main_blur_edge_detect(grid_32_32, block_32_32, shared, streams, rgb_in, rgb_tmp_blur_edge_detect, rgb_out_blur_edge_detect, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    main_edge_detect_blur(grid_32_32, block_32_32, shared, streams, rgb_in, rgb_tmp_edge_detect_blur, rgb_out_edge_detect_blur, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    // Recup donnees kernel
    for (std::size_t i = 0; i < taille_stream; ++i) {
        err = hipMemcpyAsync(g_blur.data() + i * taille_rgb / taille_stream,
                              rgb_out_blur + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
        err = hipMemcpyAsync(g_sharpen.data() + i * taille_rgb / taille_stream,
                              rgb_out_sharpen + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
        err = hipMemcpyAsync(g_edge_detect.data() + i * taille_rgb / taille_stream,
                              rgb_out_edge_detect + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

        err = hipMemcpyAsync(g_blur_edge_detect.data() + i * taille_rgb / taille_stream,
                rgb_out_blur_edge_detect + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
        err = hipMemcpyAsync(g_edge_detect_blur.data() + i * taille_rgb / taille_stream,
                rgb_out_edge_detect_blur + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    }

    hipDeviceSynchronize();

    cv::imwrite(out + std::string("_shared_stream_block_32-32_blur.") + std::string(argv[2]), m_out_blur);
    cv::imwrite(out + std::string("_shared_stream_block_32-32_sharpen.") + std::string(argv[2]), m_out_sharpen);
    cv::imwrite(out + std::string("_shared_stream_block_32-32_edge_detect.") + std::string(argv[2]), m_out_edge_detect);

    cv::imwrite(out + std::string("_shared_stream_block_32-32_blur_edge_detect.") + std::string(argv[2]), m_out_blur_edge_detect);
    cv::imwrite(out + std::string("_shared_stream_block_32-32_edge_detect_blur.") + std::string(argv[2]), m_out_edge_detect_blur);

    /////////////////////////////////////////////////////////////////
    ///////////////////// block 17 20 3 /////////////////////////////
    /////////////////////////////////////////////////////////////////

    dim3 block_17_20_3(17, 20 / taille_stream, 3); //nb de thread par bloc, max 1024
    dim3 grid_17_20_3(((cols - 1) / block_17_20_3.x + 1),
            (((rows + (taille_stream - 1) * 2) / taille_stream - 1) / block_17_20_3.y + 1)); // nb de block
    shared = 3 * block_17_20_3.x * block_17_20_3.y;

    // Execution
    main_blur(grid_17_20_3, block_17_20_3, shared, streams, rgb_in, rgb_out_blur, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    main_sharpen(grid_17_20_3, block_17_20_3, shared, streams, rgb_in, rgb_out_sharpen, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    main_edge_detect(grid_17_20_3, block_17_20_3, shared, streams, rgb_in, rgb_out_edge_detect, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    main_blur_edge_detect(grid_17_20_3, block_17_20_3, shared, streams, rgb_in, rgb_tmp_blur_edge_detect, rgb_out_blur_edge_detect, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    main_edge_detect_blur(grid_17_20_3, block_17_20_3, shared, streams, rgb_in, rgb_tmp_edge_detect_blur, rgb_out_edge_detect_blur, rows, cols);
    err = hipGetLastError();
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    // Recup donnees kernel
    for (std::size_t i = 0; i < taille_stream; ++i) {
        err = hipMemcpyAsync(g_blur.data() + i * taille_rgb / taille_stream,
                              rgb_out_blur + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
        err = hipMemcpyAsync(g_sharpen.data() + i * taille_rgb / taille_stream,
                              rgb_out_sharpen + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
        err = hipMemcpyAsync(g_edge_detect.data() + i * taille_rgb / taille_stream,
                              rgb_out_edge_detect + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

        err = hipMemcpyAsync(g_blur_edge_detect.data() + i * taille_rgb / taille_stream,
                              rgb_out_blur_edge_detect + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
        err = hipMemcpyAsync(g_edge_detect_blur.data() + i * taille_rgb / taille_stream,
                              rgb_out_edge_detect_blur + i * taille_rgb / taille_stream, taille_rgb / taille_stream,
                              hipMemcpyDeviceToHost, streams[i]);
        if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    }

    hipDeviceSynchronize();

    cv::imwrite(out + std::string("_shared_stream_block_17-20-3_blur.") + std::string(argv[2]), m_out_blur);
    cv::imwrite(out + std::string("_shared_stream_block_17-20-3_sharpen.") + std::string(argv[2]), m_out_sharpen);
    cv::imwrite(out + std::string("_shared_stream_block_17-20-3_edge_detect.") + std::string(argv[2]), m_out_edge_detect);

    cv::imwrite(out + std::string("_shared_stream_block_17-20-3_blur_edge_detect.") + std::string(argv[2]), m_out_blur_edge_detect);
    cv::imwrite(out + std::string("_shared_stream_block_17-20-3_edge_detect_blur.") + std::string(argv[2]), m_out_edge_detect_blur);

    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////

    // Nettoyage memoire
    for (std::size_t i = 0; i < taille_stream; ++i ) {
        hipStreamDestroy(streams[i]);
    }

    hipFree(rgb_in);

    hipFree(rgb_out_blur);
    hipFree(rgb_out_sharpen);
    hipFree(rgb_out_edge_detect);

    return 0;
}
