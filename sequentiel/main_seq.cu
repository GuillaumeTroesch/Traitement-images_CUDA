#include <opencv2/opencv.hpp>
#include <vector>
#include <iostream>
#include <chrono>
#include <string>

void blur(const unsigned char* rgb_in, unsigned char* rgb_out_blur, int rows, int cols) {
    for (std::size_t row = 1; row < rows - 1; ++row) {
        for (std::size_t col = 1; col < cols - 1; ++col) {
            for (std::size_t rgb = 0; rgb < 3; ++rgb)
            {
                unsigned char hg = rgb_in[3 * ((row - 1) * cols + col - 1) + rgb];
                unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
                unsigned char hd = rgb_in[3 * ((row - 1) * cols + col + 1) + rgb];
                unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
                unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
                unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
                unsigned char bg = rgb_in[3 * ((row + 1) * cols + col - 1) + rgb];
                unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
                unsigned char bd = rgb_in[3 * ((row + 1) * cols + col + 1) + rgb];

                rgb_out_blur[3 * (row * cols + col) + rgb] = (hg + h + hd + g + c + d + bg + b + bd) / 9;
            }
        }
    }
}

void sharpen(const unsigned char* rgb_in, unsigned char* rgb_out_sharpen, int rows, int cols) {
    for (std::size_t row = 1; row < rows - 1; ++row) {
        for (std::size_t col = 1; col < cols - 1; ++col) {
            for (std::size_t rgb = 0; rgb < 3; ++rgb)
            {
                unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
                unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
                unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
                unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
                unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
                int somme = (-3 * (h + g + d + b) + 21 * c) / 9;

                if (somme > 255) somme = 255;
                if (somme < 0) somme = 0;

                rgb_out_sharpen[3 * (row * cols + col) + rgb] = somme;
            }
        }
    }
}

void edge_detect(const unsigned char* rgb_in, unsigned char* rgb_out_edge_detect, int rows, int cols) {
    for (std::size_t row = 1; row < rows - 1; ++row) {
        for (std::size_t col = 1; col < cols - 1; ++col) {
            for (std::size_t rgb = 0; rgb < 3; ++rgb)
            {
                unsigned char h = rgb_in[3 * ((row - 1) * cols + col) + rgb];
                unsigned char g = rgb_in[3 * (row * cols + col - 1) + rgb];
                unsigned char c = rgb_in[3 * (row * cols + col) + rgb];
                unsigned char d = rgb_in[3 * (row * cols + col + 1) + rgb];
                unsigned char b = rgb_in[3 * ((row + 1) * cols + col) + rgb];
                int somme = (9 * (h + g + d + b) - 36 * c) / 9;

                if (somme > 255) somme = 255;
                if (somme < 0) somme = 0;

                rgb_out_edge_detect[3 * (row * cols + col) + rgb] = somme;
            }
        }
    }
}


void main_blur(const unsigned char* rgb_in, unsigned char* rgb_out_blur, int rows, int cols) {
    //Debut de chrono
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    blur(rgb_in, rgb_out_blur, rows, cols);

    //Fin de chrono
    end = std::chrono::system_clock::now();
    int elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    std::cout << "blur_seq: " << elapsedTime << std::endl;
}

void main_sharpen(const unsigned char* rgb_in, unsigned char* rgb_out_sharpen, int rows, int cols)
{
    //Debut de chrono
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    sharpen(rgb_in, rgb_out_sharpen, rows, cols);

    //Fin de chrono
    end = std::chrono::system_clock::now();
    int elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
    std::cout << "sharpen_seq: " << elapsedTime << std::endl;
}

void main_edge_detect(const unsigned char* rgb_in, unsigned char* rgb_out_edge_detect, int rows, int cols)
{
    //Debut de chrono
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    edge_detect(rgb_in, rgb_out_edge_detect, rows, cols);

    //Fin de chrono
    end = std::chrono::system_clock::now();
    int elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
    std::cout << "edge_detect_seq: " << elapsedTime << std::endl;
}


void main_blur_edge_detect(const unsigned char* rgb_in, unsigned char* rgb_tmp_blur_edge_detect,
                           unsigned char* rgb_out_blur_edge_detect, int rows, int cols)
{
    //Debut de chrono
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    blur(rgb_in, rgb_tmp_blur_edge_detect, rows, cols);
    edge_detect(rgb_tmp_blur_edge_detect, rgb_out_blur_edge_detect, rows, cols);

    //Fin de chrono
    end = std::chrono::system_clock::now();
    int elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
    std::cout << "blur_edge_detect_seq: " << elapsedTime << std::endl;
}

void main_edge_detect_blur(const unsigned char* rgb_in, unsigned char* rgb_tmp_edge_detect_blur,
                           unsigned char* rgb_out_edge_detect_blur, int rows, int cols)
{
    //Debut de chrono
    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();

    edge_detect(rgb_in, rgb_tmp_edge_detect_blur, rows, cols);
    blur(rgb_tmp_edge_detect_blur, rgb_out_edge_detect_blur, rows, cols);

    //Fin de chrono
    end = std::chrono::system_clock::now();
    int elapsedTime = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
    std::cout << "edge_detect_blur_seq: " << elapsedTime << std::endl;
}


int main(int argc, char *argv[])
{
    // Declarations
    hipError_t err;

    std::string filename = std::string(argv[1]) + std::string(".") + std::string(argv[2]);
    std::string out(argv[1]);
    if (out == "in") {
        out = std::string("out");
    }

    cv::Mat m_in = cv::imread(filename, cv::IMREAD_UNCHANGED);
    auto rgb = m_in.data;
    auto rows = m_in.rows;
    auto cols = m_in.cols;

    size_t taille_rgb = 3 * rows * cols;
    std::vector< unsigned char > g_blur(taille_rgb);

    std::vector< unsigned char > g_sharpen(taille_rgb);
    std::vector< unsigned char > g_edge_detect(taille_rgb);

    std::vector< unsigned char > g_blur_edge_detect(taille_rgb);
    std::vector< unsigned char > g_edge_detect_blur(taille_rgb);

    cv::Mat m_out_blur(rows, cols, CV_8UC3, g_blur.data());

    cv::Mat m_out_sharpen(rows, cols, CV_8UC3, g_sharpen.data());
    cv::Mat m_out_edge_detect(rows, cols, CV_8UC3, g_edge_detect.data());

    cv::Mat m_out_blur_edge_detect(rows, cols, CV_8UC3, g_blur_edge_detect.data());
    cv::Mat m_out_edge_detect_blur(rows, cols, CV_8UC3, g_edge_detect_blur.data());

    unsigned char* rgb_in = nullptr;

    unsigned char* rgb_out_blur = nullptr;
    unsigned char* rgb_out_sharpen = nullptr;
    unsigned char* rgb_out_edge_detect = nullptr;

    unsigned char* rgb_tmp_blur_edge_detect = nullptr;
    unsigned char* rgb_tmp_edge_detect_blur = nullptr;
    unsigned char* rgb_out_blur_edge_detect = nullptr;
    unsigned char* rgb_out_edge_detect_blur = nullptr;

    // Init donnes kernel
    err = hipHostMalloc(&rgb_in, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    err = hipHostMalloc(&rgb_out_blur, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipHostMalloc(&rgb_out_sharpen, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipHostMalloc(&rgb_out_edge_detect, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    err = hipHostMalloc(&rgb_tmp_blur_edge_detect, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipHostMalloc(&rgb_tmp_edge_detect_blur, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipHostMalloc(&rgb_out_blur_edge_detect, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipHostMalloc(&rgb_out_edge_detect_blur, taille_rgb);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    err = hipMemcpy(rgb_in, rgb, taille_rgb, hipMemcpyHostToDevice);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    // Execution
    main_blur(rgb_in, rgb_out_blur, rows, cols);
    main_sharpen(rgb_in, rgb_out_sharpen, rows, cols);
    main_edge_detect(rgb_in, rgb_out_edge_detect, rows, cols);

    main_blur_edge_detect(rgb_in, rgb_tmp_blur_edge_detect, rgb_out_blur_edge_detect, rows, cols);
    main_edge_detect_blur(rgb_in, rgb_tmp_edge_detect_blur, rgb_out_edge_detect_blur, rows, cols);

    // Recup donnees kernel
    err = hipMemcpy(g_blur.data(), rgb_out_blur, taille_rgb, hipMemcpyDeviceToHost);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMemcpy(g_sharpen.data(), rgb_out_sharpen, taille_rgb, hipMemcpyDeviceToHost);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMemcpy(g_edge_detect.data(), rgb_out_edge_detect, taille_rgb, hipMemcpyDeviceToHost);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    err = hipMemcpy(g_blur_edge_detect.data(), rgb_out_blur_edge_detect, taille_rgb, hipMemcpyDeviceToHost);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }
    err = hipMemcpy(g_edge_detect_blur.data(), rgb_out_edge_detect_blur, taille_rgb, hipMemcpyDeviceToHost);
    if ( err != hipSuccess ) { std::cerr << "Error" << std::endl; }

    cv::imwrite(out + std::string("_seq_blur.") + std::string(argv[2]), m_out_blur);
    cv::imwrite(out + std::string("_seq_sharpen.") + std::string(argv[2]), m_out_sharpen);
    cv::imwrite(out + std::string("_seq_edge_detect.") + std::string(argv[2]), m_out_edge_detect);

    cv::imwrite(out + std::string("_seq_blur_edge_detect.") + std::string(argv[2]), m_out_blur_edge_detect);
    cv::imwrite(out + std::string("_seq_edge_detect_blur.") + std::string(argv[2]), m_out_edge_detect_blur);

    // Nettoyage memoire
    hipHostFree(rgb_in);

    hipHostFree(rgb_out_blur);
    hipHostFree(rgb_out_sharpen);
    hipHostFree(rgb_out_edge_detect);

    hipHostFree(rgb_tmp_blur_edge_detect);
    hipHostFree(rgb_tmp_edge_detect_blur);
    hipHostFree(rgb_out_blur_edge_detect);
    hipHostFree(rgb_out_edge_detect_blur);
    return 0;
}